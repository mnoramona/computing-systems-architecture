#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>


// Function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(BYTE *block_content, BYTE *block_hash, uint32_t *yes_nonce) {

    BYTE difficulty_5_zeros[SHA256_HASH_SIZE] = "0000099999999999999999999999999999999999999999999999999999999999";
    
    // Copy block_content and block_hash
    BYTE block_hash_copy[SHA256_HASH_SIZE];
    BYTE block_copy_content[BLOCK_SIZE];

    // Nonce string
    char nonce_string[NONCE_SIZE];

    // Thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride
    int stride = blockDim.x * gridDim.x;

    for (uint32_t i_nonce = index; i_nonce <= MAX_NONCE; i_nonce += stride) {
        // Append nonce to block_content
        d_strcpy((char *)block_copy_content, (const char *)block_content);
        
        // Convert nonce to string
        intToString(i_nonce, nonce_string);

        // Append nonce to block_content (copy)
        d_strcpy((char *)block_copy_content + d_strlen((const char*)block_copy_content), nonce_string);
        
        // Apply SHA256 to block_content (copy)
        apply_sha256(block_copy_content, d_strlen((const char*)block_copy_content), block_hash_copy, 1);
        
        if (compare_hashes(block_hash_copy, difficulty_5_zeros) <= 0) {
            *yes_nonce = i_nonce;  // Store nonce
            d_strcpy((char *)block_hash, (const char *)block_hash_copy);  // Store block_hash
            break;
        }
    }
}

int main(int argc, char **argv) {
    ///// Neschimbat
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
            tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
            tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE]; 
    // Top hash 
    apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

    // prev_block_hash + top_hash
    strcpy((char*)block_content, (const char*)prev_block_hash);
    strcat((char*)block_content, (const char*)top_hash);
    ///// 

    BYTE *d_block_content, *d_block_hash;
    uint32_t *d_nonce;

    // Allocate memory on the device for block_content, block_hash, and nonce
    hipMalloc((void **)&d_block_content, BLOCK_SIZE * sizeof(BYTE));
    hipMalloc((void **)&d_block_hash, (SHA256_HASH_SIZE + 1) * sizeof(BYTE));
    hipMalloc((void **)&d_nonce, sizeof(uint32_t));

    // Copy block_content to device
    hipMemcpy(d_block_content, block_content, BLOCK_SIZE * sizeof(BYTE), hipMemcpyHostToDevice);

    // Copy block_hash and nonce back to host
    BYTE *block_hash;
    block_hash = (BYTE *)malloc((SHA256_HASH_SIZE + 1) * sizeof(BYTE));

    int blockSize = 256;
    int numBlocks = (MAX_NONCE + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    startTiming(&start, &stop);

    // Launch kernel
    findNonce<<<numBlocks, blockSize>>>(d_block_content, d_block_hash, d_nonce);
    hipDeviceSynchronize();

    float seconds = stopTiming(&start, &stop);

    uint32_t nonce = 0;
    
    hipMemcpy(&nonce, d_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(block_hash, d_block_hash, SHA256_HASH_SIZE * sizeof(BYTE), hipMemcpyDeviceToHost);

    printResult(block_hash, nonce, seconds);

    // Free device memory
    hipFree(d_block_content);
    hipFree(d_block_hash);
    hipFree(d_nonce);

    // Free host memory
    free(block_hash);

    return 0;
}